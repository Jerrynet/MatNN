#include "utils.h"
/***
 PRHS[0] = ActivationMode
 PRHS[1] = input data 
 PRHS[2] = output data (Format in Matlab = HWCN)
 PRHS[3] = output diff (Format in Matlab = HWCN)
 PLHS[0] = input diff (Format in Matlab = HWCN)
 ------------
 int ActivationMode, 0 = HIPDNN_ACTIVATION_SIGMOID
                     1 = HIPDNN_ACTIVATION_RELU
                     2 = HIPDNN_ACTIVATION_TANH
                     3 = HIPDNN_ACTIVATION_CLIPPED_RELU
 */
hipdnnHandle_t cudnnHandle;
bool initialized = false;
void destroyHandle(){
  CUDNN_CHECK( hipdnnDestroy(cudnnHandle) );
}
void mexFunction( int nlhs,       mxArray *plhs[], 
                  int nrhs, const mxArray *prhs[]  )
{
    // Check nargin / nargout
    if (nrhs != 4) mexErrMsgIdAndTxt("MATLAB:MatNN", "Accepts 4 inputs."); // mode, input, output, output_diff
    if (nlhs != 1) mexErrMsgIdAndTxt("MATLAB:MatNN", "Accepts 1 output.");

    // Check prhs[0] == int
    hipdnnActivationMode_t mode = (hipdnnActivationMode_t)mxGetScalar(prhs[0]);

    // Get inputs / outputs data
    mxGPUArray const *input = mxGPUCreateFromMxArray(prhs[1]);
    gpuData inputInfo = createInfoFromMxGPUArray(input);
    float *input_data = (float *)mxGPUGetDataReadOnly(input);

    mxGPUArray const *out = mxGPUCreateFromMxArray(prhs[2]);
    gpuData out_info = createInfoFromMxGPUArray(out);
    float *out_data = (float *)mxGPUGetDataReadOnly(out);

    mxGPUArray const *out_diff = mxGPUCreateFromMxArray(prhs[3]);
    gpuData out_diff_info = createInfoFromMxGPUArray(out_diff);
    float *out_diff_data = (float *)mxGPUGetDataReadOnly(out_diff);

    mxGPUArray *in_diff = mxGPUCreateGPUArray(inputInfo.ndims,
                                inputInfo.dims,
                                inputInfo.type,
                                mxREAL,
                                MX_GPU_DO_NOT_INITIALIZE); // MX_GPU_INITIALIZE_VALUES
    float *in_diff_data = (float *)mxGPUGetData(in_diff);

    // init
    float alpha = 1.0;
    float beta  = 0.0;
    
    hipdnnTensorDescriptor_t srcTensorDesc, srcDiffTensorDesc, dstTensorDesc, dstDiffTensorDesc;
    if (!initialized){
      CUDNN_CHECK( hipdnnCreate(&cudnnHandle) );
      mexAtExit(destroyHandle);
      initialized = true;
    }
    CUDNN_CHECK( hipdnnCreateTensorDescriptor(&srcTensorDesc) );
    CUDNN_CHECK( hipdnnCreateTensorDescriptor(&srcDiffTensorDesc) );
    CUDNN_CHECK( hipdnnCreateTensorDescriptor(&dstTensorDesc) );
    CUDNN_CHECK( hipdnnCreateTensorDescriptor(&dstDiffTensorDesc) );
    

    // Set sizes
    CUDNN_CHECK( hipdnnSetTensor4dDescriptorEx(srcTensorDesc,
                                              HIPDNN_DATA_FLOAT,
                                              inputInfo.n, inputInfo.c, inputInfo.h, inputInfo.w,
                                              inputInfo.nStride, inputInfo.cStride, inputInfo.hStride, inputInfo.wStride) );
    CUDNN_CHECK( hipdnnSetTensor4dDescriptorEx(srcDiffTensorDesc,
                                              HIPDNN_DATA_FLOAT,
                                              inputInfo.n, inputInfo.c, inputInfo.h, inputInfo.w,
                                              inputInfo.nStride, inputInfo.cStride, inputInfo.hStride, inputInfo.wStride) );
    CUDNN_CHECK( hipdnnSetTensor4dDescriptorEx(dstDiffTensorDesc,
                                              HIPDNN_DATA_FLOAT,
                                              inputInfo.n, inputInfo.c, inputInfo.h, inputInfo.w,
                                              inputInfo.nStride, inputInfo.cStride, inputInfo.hStride, inputInfo.wStride) );
    CUDNN_CHECK( hipdnnSetTensor4dDescriptorEx(dstTensorDesc,
                                              HIPDNN_DATA_FLOAT,
                                              inputInfo.n, inputInfo.c, inputInfo.h, inputInfo.w,
                                              inputInfo.nStride, inputInfo.cStride, inputInfo.hStride, inputInfo.wStride) );
    

    // compute
    CUDNN_CHECK( hipdnnActivationBackward(cudnnHandle,
                                        mode,
                                        &alpha,
                                        dstTensorDesc,
                                        out_data,
                                        dstDiffTensorDesc,
                                        out_diff_data,
                                        srcTensorDesc,
                                        input_data,
                                        &beta,
                                        srcDiffTensorDesc,
                                        in_diff_data) );

    // Set outputs
    //hipdnnDestroy(cudnnHandle);
    CUDNN_CHECK( hipdnnDestroyTensorDescriptor(srcTensorDesc) );
    CUDNN_CHECK( hipdnnDestroyTensorDescriptor(srcDiffTensorDesc) );
    CUDNN_CHECK( hipdnnDestroyTensorDescriptor(dstTensorDesc) );
    CUDNN_CHECK( hipdnnDestroyTensorDescriptor(dstDiffTensorDesc) );
    mxGPUDestroyGPUArray(input);
    mxGPUDestroyGPUArray(out_diff);
    mxGPUDestroyGPUArray(out);
    plhs[0] = mxGPUCreateMxArrayOnGPU(in_diff);
    mxGPUDestroyGPUArray(in_diff);
}
#include "utils.h"
/***
 PRHS[0] = ActivationMode
 PRHS[1] = input data (Format in Matlab = HWCN)
 PLHS[0] = output data (Format in Matlab = HWCN)
 ------------
 int ActivationMode, 0 = HIPDNN_ACTIVATION_SIGMOID
                     1 = HIPDNN_ACTIVATION_RELU
                     2 = HIPDNN_ACTIVATION_TANH
                     3 = HIPDNN_ACTIVATION_CLIPPED_RELU
 */
hipdnnHandle_t cudnnHandle;
bool initialized = false;
void destroyHandle(){
  CUDNN_CHECK( hipdnnDestroy(cudnnHandle) );
}
void mexFunction( int nlhs,       mxArray *plhs[], 
                  int nrhs, const mxArray *prhs[]  )
{
    // Check nargin / nargout
    if (nrhs != 2) mexErrMsgIdAndTxt("MATLAB:MatNN", "Accepts 2 inputs.");
    if (nlhs != 1) mexErrMsgIdAndTxt("MATLAB:MatNN", "Accepts 1 output.");

    // Check prhs[0] == int
    hipdnnActivationMode_t mode = (hipdnnActivationMode_t)mxGetScalar(prhs[0]);

    // Get inputs / outputs data
    mxGPUArray const *input1 = mxGPUCreateFromMxArray(prhs[1]);
    gpuData input1Info = createInfoFromMxGPUArray(input1);
    float *input1_data = (float *)mxGPUGetDataReadOnly(input1);

    
    mxGPUArray *output1 = mxGPUCreateGPUArray(input1Info.ndims,
                                input1Info.dims,
                                input1Info.type,
                                mxREAL,
                                MX_GPU_DO_NOT_INITIALIZE); // MX_GPU_INITIALIZE_VALUES
    float *output1_data = (float *)mxGPUGetData(output1);

    // init
    float alpha = 1.0;
    float beta  = 0.0;
    
    hipdnnTensorDescriptor_t srcTensorDesc, dstTensorDesc;
    if (!initialized){
      CUDNN_CHECK( hipdnnCreate(&cudnnHandle) );
      mexAtExit(destroyHandle);
      initialized = true;
    }
    CUDNN_CHECK( hipdnnCreateTensorDescriptor(&srcTensorDesc) );
    CUDNN_CHECK( hipdnnCreateTensorDescriptor(&dstTensorDesc) );

    // Set sizes
    CUDNN_CHECK( hipdnnSetTensor4dDescriptorEx(srcTensorDesc,
                                              HIPDNN_DATA_FLOAT,
                                              input1Info.n, input1Info.c, input1Info.h, input1Info.w,
                                              input1Info.nStride, input1Info.cStride, input1Info.hStride, input1Info.wStride) );
    CUDNN_CHECK( hipdnnSetTensor4dDescriptorEx(dstTensorDesc,
                                              HIPDNN_DATA_FLOAT,
                                              input1Info.n, input1Info.c, input1Info.h, input1Info.w,
                                              input1Info.nStride, input1Info.cStride, input1Info.hStride, input1Info.wStride) );

    // compute
    CUDNN_CHECK( hipdnnActivationForward(cudnnHandle,
                                        mode,
                                        &alpha,
                                        srcTensorDesc,
                                        input1_data,
                                        &beta,
                                        dstTensorDesc,
                                        output1_data) );

    // Set outputs
    //hipdnnDestroy(cudnnHandle);
    CUDNN_CHECK( hipdnnDestroyTensorDescriptor(srcTensorDesc) );
    CUDNN_CHECK( hipdnnDestroyTensorDescriptor(dstTensorDesc) );
    mxGPUDestroyGPUArray(input1);
    plhs[0] = mxGPUCreateMxArrayOnGPU(output1);
    mxGPUDestroyGPUArray(output1);
}